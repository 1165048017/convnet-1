#include "hip/hip_runtime.h"
/** Kernels for convUp, convDown, convOutp, maxpool, avgpool, maxpoolundo,
 *  avgpoolundo.
 *  These kernels are 10-20% slower than cuda-convnet2, but have no constraints
 *  on number of channels and support rectangular images and rectangular kernels.
 *  They use hipblasSgemm for convUp, convDown, convOutp.
 *  Data layout : Column-major
 *  data : (num_images, image_size_x, image_size_y, num_input_channels)
 *  filters : (num_output_channels, kernel_size_x, kernel_size_y, num_input_channels)
 */

#include "cudamat_conv_gemm.cuh"
#define getLastCudaError(msg)   __getLastCudaError (msg, __FILE__, __LINE__)

inline bool check_cublas_error() {
  hipblasStatus_t status = cublasGetError();
  return status != HIPBLAS_STATUS_SUCCESS;
}

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line) {
 hipError_t err = hipGetLastError();
 if (hipSuccess != err) {
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString(err));
  exit(EXIT_FAILURE);
 }
}

class AvgPooler {
 public:
  __device__ inline float operator()(const float a, const float b) const {
    return a + b;
  }
  __device__ inline float getBaseValue() const {
    return 0;
  }
  __device__ inline float output(const float a, const int regionSize) const {
    return a / regionSize;
  }
};

class MaxPooler {
 public:
  __device__ inline float operator()(const float a, const float b) const {
    return fmaxf(a, b);
  }
  __device__ inline float getBaseValue() const {
    return -2e38; 
  }
  __device__ inline float output(const float a, const int regionSize) const {
    return a;
  }
};

__global__ void kExpand(float *images, float* targets,
                        int num_images, int num_input_channels,
                        int image_size_y, int image_size_x,
                        int num_modules_y, int num_modules_x,
                        int kernel_size_y, int kernel_size_x,
                        int padding_y, int padding_x,
                        int stride_y, int stride_x,
                        int num_modules_batch, int module_id_offset) {
  int color = blockIdx.y;
  int src_module_id = module_id_offset + blockIdx.x;
  int dst_module_id = blockIdx.x;

  int module_id_x = src_module_id % num_modules_x;
  int module_id_y = src_module_id / num_modules_x;
  int startX = module_id_x * stride_x + padding_x;
  int startY = module_id_y * stride_y + padding_y;
  int Y, X;
  long target_id, source_id;
  images += num_images * image_size_x * image_size_y * color;
  targets += num_images * (dst_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
  for (int y = 0; y < kernel_size_y; y++) {
    Y = startY + y;
    for (int x = 0; x < kernel_size_x; x++) {
      X = startX + x;
      target_id = num_images * num_modules_batch * (x + kernel_size_x * y);
      source_id = num_images * (X + image_size_x * Y);
      if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          targets[target_id + im] = 0;
        }
      } else {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          targets[target_id + im] = images[source_id + im];
        }
      }
      __syncthreads();
    }
  }
}

template <class Pooler>
__global__ void kPool(float *images, float* targets,
                      int num_images, int num_input_channels,
                      int image_size_y, int image_size_x,
                      int num_modules_y, int num_modules_x,
                      int kernel_size_y, int kernel_size_x,
                      int padding_y, int padding_x,
                      int stride_y, int stride_x, float scaleOutput,
                      Pooler pooler) {
  int color = blockIdx.y;
  int num_modules = num_modules_y * num_modules_x;

  long source_id, target_id;
  images += num_images * image_size_x * image_size_y * color;
  targets += num_images * num_modules * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    target_id = num_images * module_id;
    int endY = startY + kernel_size_y;
    int endX = startX + kernel_size_x;
    startY = MAX(startY, 0);
    startX = MAX(startX, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    int regionSize = (endX - startX) * (endY - startY);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = pooler.getBaseValue();
      for (int Y = startY; Y < endY; Y++) {
        for (int X = startX; X < endX; X++) {
          source_id = num_images * (X + image_size_x * Y);
          val = pooler(val, images[source_id + im]);
        }
      }
      targets[target_id + im] = scaleOutput * pooler.output(val, regionSize);
    }
  }
  __syncthreads();
}

__global__ void kAvgPoolUndo(float *derivs, float* targets,
                             int num_images, int num_input_channels,
                             int image_size_y, int image_size_x,
                             int num_modules_y, int num_modules_x,
                             int kernel_size_y, int kernel_size_x,
                             int padding_y, int padding_x,
                             int stride_y, int stride_x, float scaleOutput) {
  int color = blockIdx.y;
  int num_modules = num_modules_y * num_modules_x;

  long source_id;
  derivs  += num_images * num_modules * color;
  targets += num_images * image_size_x * image_size_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    source_id = num_images * module_id;
    int endY = startY + kernel_size_y;
    int endX = startX + kernel_size_x;
    startY = MAX(startY, 0);
    startX = MAX(startX, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    int regionSize = (endX - startX) * (endY - startY);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = scaleOutput * derivs[source_id + im] / regionSize;
      for (int Y = startY; Y < endY; Y++) {
        for (int X = startX; X < endX; X++) {
          atomicAdd(&targets[num_images * (X + image_size_x * Y) + im], val);
          __syncthreads();
        }
      }
    }
  }
}

__global__ void kMaxPoolUndo(float * images, float *derivs, float* maxes, float* targets,
                        int num_images, int num_input_channels,
                        int image_size_y, int image_size_x,
                        int num_modules_y, int num_modules_x,
                        int kernel_size_y, int kernel_size_x,
                        int padding_y, int padding_x,
                        int stride_y, int stride_x, float scaleOutput) {
  int color = blockIdx.y;
  int num_modules = num_modules_y * num_modules_x;

  long source_id, target_id;
  derivs  += num_images * num_modules * color;
  maxes  += num_images * num_modules * color;
  targets += num_images * image_size_x * image_size_y * color;
  images += num_images * image_size_x * image_size_y * color;
  for (int module_id = blockIdx.x; module_id < num_modules; module_id += gridDim.x) {
    int module_id_x = module_id % num_modules_x;
    int module_id_y = module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    source_id = num_images * module_id;
    int endY = startY + kernel_size_y;
    int endX = startX + kernel_size_x;
    startY = MAX(startY, 0);
    startX = MAX(startX, 0);
    endY   = MIN(endY  , image_size_y);
    endX   = MIN(endX  , image_size_x);
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
      float val = scaleOutput * derivs[source_id + im];
      for (int Y = startY; Y < endY; Y++) {
        for (int X = startX; X < endX; X++) {
          target_id = num_images * (X + image_size_x * Y) + im;
          if (images[target_id] == maxes[source_id + im]) {
            atomicAdd(&targets[target_id], val);
          }
          __syncthreads();
        }
      }
    }
  }
}

__global__ void kContract(float *expanded_data, float* targets,
                          int num_images, int num_input_channels,
                          int image_size_y, int image_size_x,
                          int num_modules_y, int num_modules_x,
                          int kernel_size_y, int kernel_size_x,
                          int padding_y, int padding_x,
                          int stride_y, int stride_x,
                          int num_modules_batch, int module_id_offset) {
  int color = blockIdx.y;
  int dst_module_id = module_id_offset + blockIdx.x;
  int src_module_id = blockIdx.x;

  int module_id_x = dst_module_id % num_modules_x;
  int module_id_y = dst_module_id / num_modules_x;
  int startX = module_id_x * stride_x + padding_x;
  int startY = module_id_y * stride_y + padding_y;
  int Y, X;
  long target_id, source_id;
  targets += num_images * image_size_x * image_size_y * color;
  expanded_data  += num_images * (src_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
  for (int y = 0; y < kernel_size_y; y++) {
    Y = startY + y;
    for (int x = 0; x < kernel_size_x; x++) {
      X = startX + x;
      source_id = num_images * num_modules_batch * (x + kernel_size_x * y);
      target_id = num_images * (X + image_size_x * Y);
      if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
        // do nothing.
      } else {
        for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
          atomicAdd(&targets[target_id + im], expanded_data[source_id + im]);
          __syncthreads();
        }
      }
    }
  }
}

__global__ void kWriteRows(float* data, float* target,
                               int num_images, int num_modules,
                               int num_modules_batch, int module_id_offset,
                               float beta) {
  int c = blockIdx.y;
  int src_module_id = blockIdx.x;
  int dst_module_id = module_id_offset + blockIdx.x;

  data += num_images * (src_module_id + c * num_modules_batch);
  target += num_images * (dst_module_id + c * num_modules);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = beta * data[im];
  }
}

__global__ void kReadRows(float* data, float* target,
                          int num_images, int num_modules,
                          int num_modules_batch, int module_id_offset) {
  int c = blockIdx.y;
  int src_module_id = module_id_offset + blockIdx.x;
  int dst_module_id = blockIdx.x;

  data += num_images * (src_module_id + c * num_modules);
  target += num_images * (dst_module_id + c * num_modules_batch);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = data[im];
  }
}


__global__ void kWriteRowsMult(float* data, float* target,
                               int num_images, int num_modules,
                               int num_modules_batch, int module_id_offset,
                               float alpha, float beta) {
  int c = blockIdx.y;
  int src_module_id = blockIdx.x;
  int dst_module_id = module_id_offset + blockIdx.x;

  data += num_images * (src_module_id + c * num_modules_batch);
  target += num_images * (dst_module_id + c * num_modules);

  for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
    target[im] = alpha * target[im] + beta * data[im];
  }
}

__global__ void kCrossMapDenoms(float* data, float* denoms,
                                int num_locs, int batch_locs, int batch_offset, float addScale,
                                int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  data   += batch_offset + loc_id;
  denoms += loc_id;
  if (batch_offset + loc_id < num_locs) {
    for (int j = 0; j < num_filters; j++) {
      float sum = 0;
      int start = blocked ? (j / k) * k : -k/2 + j;
      int end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = start; i < end; i++) {
        sum += data[i * num_locs] * data[i * num_locs];
      }
      denoms[j * batch_locs] = 1 + addScale * sum;
    }
  }
}

__global__ void kCrossMapRNorm(float* data, float* target,
                               int num_locs, float addScale, float powScale,
                               int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  data   += loc_id;
  target += loc_id;
  if (loc_id < num_locs) {
    for (int j = 0; j < num_filters; j++) {
      float sum = 0;
      int start = blocked ? (j / k) * k : -k/2 + j;
      int end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = start; i < end; i++) {
        sum += data[i * num_locs] * data[i * num_locs];
      }
      target[j * num_locs] = data[j * num_locs] * __powf(1 + addScale * sum, -powScale);
    }
  }
}

__global__ void kCrossMapRNormUndo(float* data, float* deriv, float* denoms, float* target,
                                   int num_locs, int batch_locs, int batch_offset, float addScale, float powScale,
                                   int num_filters, int k, bool blocked) {
  long loc_id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  data   += batch_offset + loc_id;
  target += batch_offset + loc_id;
  deriv  += batch_offset + loc_id;
  denoms += loc_id;
  if (batch_offset + loc_id < num_locs) {
    for (int j = 0; j < num_filters; j++) {
      float sum = 0;
      int start = blocked ? (j / k) * k : -k + k/2 + j + 1;
      int end = MIN(num_filters, start + k);
      start = MAX(0, start);
      for (int i = start; i < end; i++) {
        sum += deriv[i * num_locs] * data[i * num_locs] * __powf(denoms[i * batch_locs], -powScale - 1);
      }
      target[j * num_locs] = deriv[j * num_locs] * __powf(denoms[j * batch_locs], -powScale) -
                             2 * addScale * powScale * data[j * num_locs] * sum;
    }
  }
}

void _convUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D images_shape, Shape4D filters_shape,
                Shape4D targets_shape, ConvDesc conv_desc,
                float scaleTargets, float scaleOutput, bool conv) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int num_groups           = conv_desc.num_groups;

    int num_output_channels2 = targets_shape.shape[3];
    int num_modules_y        = targets_shape.shape[2];
    int num_modules_x        = targets_shape.shape[1];
    int num_images           = targets_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_input_channels3  = filters_shape.shape[3];
    int kernel_size_y2       = filters_shape.shape[2];
    int kernel_size_x2       = filters_shape.shape[1];
    int num_output_channels3 = filters_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int input_size           = kernel_size_y * kernel_size_x * num_input_channels;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_output_channels == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (num_input_channels == num_input_channels3 / filterModuleMult);
    assert (num_images == images->size[0]);
    assert (num_images == targets->size[0]);
    assert (num_output_channels == filters->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == targets->size[1]);
    assert (kernel_size_y * kernel_size_x * num_input_channels * filterModuleMult == filters->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);

    // Batchsize be multiple of 128 for max utilization, will still work if is isn't.
    int num_threads_x = MIN(num_images, 128);
    
    float *expanded_images = NULL, *expanded_target = NULL;
    int num_modules_batch;
    
    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    int max_batch_size = ((long) MAX_MEMORY_BYTES) / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, 4096);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_target, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      if (hipSuccess == err1) hipFree(expanded_images);
      if (hipSuccess == err2) hipFree(expanded_target);
      err1 = hipMalloc((void**)&expanded_images,  input_memory_size);
      err2 = hipMalloc((void**)&expanded_target, output_memory_size);
      if (hipSuccess != err1 || hipSuccess != err2) {
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err1));
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err2));
      } 
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);

    int module_id_start = 0;
    float* w = filters->data_device;
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 threads(num_threads_x);
      dim3 blocks = dim3(this_num_modules_batch, num_input_channels);
      kExpand<<<blocks, threads>>>(images->data_device, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      if (!conv) w += num_output_channels * input_size;
      hipblasSgemm('n', 't', 
                  num_images * this_num_modules_batch, num_output_channels,
                  kernel_size_x * kernel_size_y * num_input_channels,
                  1, expanded_images, num_images * this_num_modules_batch,
                  w, num_output_channels,
                  0, expanded_target, num_images * this_num_modules_batch);

      dim3 blocks2 = dim3(this_num_modules_batch, num_output_channels);
      if (scaleTargets == 0) {
        kWriteRows<<<blocks2, threads>>>(expanded_target, targets->data_device,
                                         num_images, num_modules,
                                         this_num_modules_batch, module_id_start,
                                         scaleOutput);
      } else {
        kWriteRowsMult<<<blocks2, threads>>>(expanded_target, targets->data_device,
                                         num_images, num_modules,
                                         this_num_modules_batch, module_id_start,
                                         scaleTargets, scaleOutput);
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_images);
    hipFree(expanded_target);
    getLastCudaError("convUpGemm: kernel execution failed");
}

void _convDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
                Shape4D derivs_shape, Shape4D filters_shape,
                Shape4D targets_shape, ConvDesc conv_desc,
                float scaleTargets, float scaleOutput, bool conv) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int num_groups           = conv_desc.num_groups;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int num_input_channels3  = filters_shape.shape[3];
    int kernel_size_y2       = filters_shape.shape[2];
    int kernel_size_x2       = filters_shape.shape[1];
    int num_output_channels3 = filters_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int input_size           = kernel_size_y * kernel_size_x * num_input_channels;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_output_channels == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (num_input_channels == num_input_channels3 / filterModuleMult);
    assert (num_images == targets->size[0]);
    assert (num_images == derivs->size[0]);
    assert (num_output_channels == filters->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == targets->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);
    assert (kernel_size_y * kernel_size_x * num_input_channels * filterModuleMult == filters->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);

    int num_threads_x = MIN(num_images, 128); // Batchsize be multiple of 128 for max utilization, will still work if is isn't.
    float *expanded_target = NULL, *expanded_derivs = NULL;
    int num_modules_batch;
    //GetTempMemory(num_images, input_size, num_output_channels, num_modules / filterModuleMult,
    //              expanded_target, expanded_derivs, &num_modules_batch);


    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    int max_batch_size = ((long) MAX_MEMORY_BYTES) / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, 4096);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_target,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_derivs, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      if (hipSuccess == err1) hipFree(expanded_target);
      if (hipSuccess == err2) hipFree(expanded_derivs);
      err1 = hipMalloc((void**)&expanded_target,  input_memory_size);
      err2 = hipMalloc((void**)&expanded_derivs, output_memory_size);
      if (hipSuccess != err1 || hipSuccess != err2) {
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err1));
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err2));
      } 
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);
    
    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    int module_id_start = 0;
    float* w = filters->data_device;
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks = dim3(this_num_modules_batch, num_output_channels);
      dim3 threads(num_threads_x);
      kReadRows<<<blocks, threads>>>(derivs->data_device, expanded_derivs,
                                     num_images, num_modules,
                                     this_num_modules_batch, module_id_start);
      if (!conv) w += num_output_channels * input_size;
      hipblasSgemm('n', 'n', 
                  num_images * this_num_modules_batch, kernel_size_x * kernel_size_y * num_input_channels,
                  num_output_channels,
                  scaleOutput, expanded_derivs, num_images * this_num_modules_batch,
                  w, num_output_channels,
                  0, expanded_target, num_images * this_num_modules_batch);

      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kContract<<<blocks2, threads>>>(expanded_target, targets->data_device,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_derivs);
    hipFree(expanded_target);
    getLastCudaError("convDownGemm: kernel execution failed");
}

void _convOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D images_shape, Shape4D derivs_shape, Shape4D targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput, bool conv) {

    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;
    int num_groups           = conv_desc.num_groups;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_input_channels3Mult  = targets_shape.shape[3];
    int kernel_size_y2       = targets_shape.shape[2];
    int kernel_size_x2       = targets_shape.shape[1];
    int num_output_channels3 = targets_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
    int input_size           = kernel_size_y * kernel_size_x * num_input_channels;
    int filterModuleMult     = conv ? 1 : num_modules;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_output_channels == num_output_channels3);
    assert (num_input_channels == num_input_channels2);
    assert (num_input_channels * filterModuleMult == num_input_channels3Mult);
    assert (num_images == images->size[0]);
    assert (num_images == derivs->size[0]);
    assert (num_output_channels == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);
    assert (kernel_size_y * kernel_size_x * num_input_channels3Mult == targets->size[1]);
    assert (kernel_size_y == kernel_size_y2);
    assert (kernel_size_x == kernel_size_x2);
    assert (num_input_channels % num_groups == 0);
    assert (num_groups == 1);

    // Batchsize be multiple of 128 for max utilization, will still work if is isn't.
    int num_threads_x = MIN(num_images, 128);
    
    float *expanded_images = NULL, *expanded_derivs = NULL;
    int num_modules_batch;
    //GetTempMemory(num_images, input_size, num_output_channels, num_modules / filterModuleMult,
    //              expanded_images, expanded_derivs, &num_modules_batch);


    int input_memory_size  = num_images * input_size * sizeof(float);
    int output_memory_size = num_images * num_output_channels * sizeof(float);
    int max_batch_size = ((long) MAX_MEMORY_BYTES) / (input_memory_size + output_memory_size);
    max_batch_size = MIN(max_batch_size, num_modules / filterModuleMult);
    max_batch_size = MIN(max_batch_size, 4096);
    max_batch_size = MAX(max_batch_size, 1);

    hipError_t err1, err2;
    err1 = hipMalloc((void**)&expanded_images,  max_batch_size * input_memory_size);
    err2 = hipMalloc((void**)&expanded_derivs, max_batch_size * output_memory_size);
    if (hipSuccess != err1 || hipSuccess != err2) {
      if (hipSuccess == err1) hipFree(expanded_images);
      if (hipSuccess == err2) hipFree(expanded_derivs);
      err1 = hipMalloc((void**)&expanded_images,  input_memory_size);
      err2 = hipMalloc((void**)&expanded_derivs, output_memory_size);
      if (hipSuccess != err1 || hipSuccess != err2) {
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err1));
        printf("Out of memory on GPU! %s \n", hipGetErrorString(err2));
      } 
      num_modules_batch = 1;
    } else {
      num_modules_batch = max_batch_size;
    }

    int num_iter = DIVUP(num_modules, num_modules_batch);

    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    int module_id_start = 0;
    dim3 threads(num_threads_x);
    float* dw = targets->data_device;
    for (int i = 0; i < num_iter; i++) {
      int this_num_modules_batch = MIN(num_modules_batch, num_modules - module_id_start);
      //printf("Step %d num_modules %d\n", i, this_num_modules_batch);

      dim3 blocks = dim3(this_num_modules_batch, num_output_channels);
      kReadRows<<<blocks, threads>>>(derivs->data_device, expanded_derivs,
                                     num_images, num_modules,
                                     this_num_modules_batch, module_id_start);
      dim3 blocks2 = dim3(this_num_modules_batch, num_input_channels);
      kExpand<<<blocks2, threads>>>(images->data_device, expanded_images,
                                   num_images, num_input_channels,
                                   image_size_y, image_size_x,
                                   num_modules_y, num_modules_x,
                                   kernel_size_y, kernel_size_x,
                                   padding_y, padding_x,
                                   stride_y, stride_x,
                                   this_num_modules_batch, module_id_start);
      if (!conv) dw += num_output_channels * input_size;
      hipblasSgemm('t', 'n', 
                  num_output_channels,
                  kernel_size_x * kernel_size_y * num_input_channels,
                  num_images * this_num_modules_batch,
                  scaleOutput, expanded_derivs, num_images * this_num_modules_batch,
                  expanded_images, num_images * this_num_modules_batch,
                  1, dw, num_output_channels);
      if (check_cublas_error()) {
        printf("Error in dot or before it.\n");
      }
      module_id_start += this_num_modules_batch;
    }
    hipFree(expanded_derivs);
    hipFree(expanded_images);
    getLastCudaError("convOutpGemm: kernel execution failed");
}

template <class Pooler>
void _convPoolGemm(cudamat* images, cudamat* targets,
                Shape4D images_shape, Shape4D targets_shape,
                ConvDesc conv_desc, float scaleTargets, float scaleOutput, Pooler pooler) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;

    int num_output_channels2 = targets_shape.shape[3];
    int num_modules_y        = targets_shape.shape[2];
    int num_modules_x        = targets_shape.shape[1];
    int num_images           = targets_shape.shape[0];

    int num_input_channels2  = images_shape.shape[3];
    int image_size_y         = images_shape.shape[2];
    int image_size_x         = images_shape.shape[1];
    int num_images2          = images_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_input_channels == num_input_channels2);
    assert (num_images == images->size[0]);
    assert (num_images == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == images->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == targets->size[1]);

    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    dim3 threads(128);
    int num_blocks_x = MIN(4096, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kPool<<<blocks, threads>>>(images->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x,
                               num_modules_y, num_modules_x,
                               kernel_size_y, kernel_size_x,
                               padding_y, padding_x,
                               stride_y, stride_x, scaleOutput, pooler);
    getLastCudaError("convLocalPool: kernel execution failed");
}

void _avgPoolUndoGemm(cudamat* derivs, cudamat* targets,
                Shape4D derivs_shape, Shape4D targets_shape,
                ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_input_channels == num_input_channels2);
    assert (num_images == derivs->size[0]);
    assert (num_images == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == targets->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);

    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    dim3 threads(128);
    int num_blocks_x = MIN(4096, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kAvgPoolUndo<<<blocks, threads>>>(derivs->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x,
                               num_modules_y, num_modules_x,
                               kernel_size_y, kernel_size_x,
                               padding_y, padding_x,
                               stride_y, stride_x, scaleOutput);
    getLastCudaError("avgPoolUndo: kernel execution failed");
}

void _maxPoolUndoGemm(cudamat* images, cudamat* derivs, cudamat* maxes, cudamat* targets,
                Shape4D targets_shape, Shape4D derivs_shape, 
                ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
    
    int num_input_channels   = conv_desc.num_input_channels;
    int num_output_channels  = conv_desc.num_output_channels;
    int kernel_size_y        = conv_desc.kernel_size_y;
    int kernel_size_x        = conv_desc.kernel_size_x;
    int stride_y             = conv_desc.stride_y;
    int stride_x             = conv_desc.stride_x;
    int padding_y            = conv_desc.padding_y;
    int padding_x            = conv_desc.padding_x;

    int num_output_channels2 = derivs_shape.shape[3];
    int num_modules_y        = derivs_shape.shape[2];
    int num_modules_x        = derivs_shape.shape[1];
    int num_images           = derivs_shape.shape[0];

    int num_input_channels2  = targets_shape.shape[3];
    int image_size_y         = targets_shape.shape[2];
    int image_size_x         = targets_shape.shape[1];
    int num_images2          = targets_shape.shape[0];

    int num_modules          = num_modules_y * num_modules_x;
  
    // Consistency checks. 
    assert (num_images == num_images2);
    assert (num_output_channels == num_output_channels2);
    assert (num_input_channels == num_input_channels2);
    assert (num_images == derivs->size[0]);
    assert (num_images == targets->size[0]);
    assert (image_size_y * image_size_x * num_input_channels == targets->size[1]);
    assert (num_modules_y * num_modules_x * num_output_channels == derivs->size[1]);

    if (scaleTargets == 0) {
      hipMemset(targets->data_device, 0, sizeof(float) * targets->size[0] * targets->size[1]);
    } else if (scaleTargets != 1) {
      hipblasSscal(sizeof(float) * targets->size[0] * targets->size[1], scaleTargets, targets->data_device, 1);
    }

    dim3 threads(128);
    int num_blocks_x = MIN(4096, num_modules);
    dim3 blocks = dim3(num_blocks_x, num_input_channels);
    kMaxPoolUndo<<<blocks, threads>>>(images->data_device, derivs->data_device,
                               maxes->data_device, targets->data_device,
                               num_images, num_input_channels,
                               image_size_y, image_size_x,
                               num_modules_y, num_modules_x,
                               kernel_size_y, kernel_size_x,
                               padding_y, padding_x,
                               stride_y, stride_x, scaleOutput);
    getLastCudaError("avgPoolUndo: kernel execution failed");
}



void _CrossMapRNorm(cudamat* images, cudamat* targets, int num_filters, int sizeF, float addScale, float powScale, bool blocked) {
  int num_locs = (images->size[0] * images->size[1]) / num_filters;
  int threads = 512;
  int num_blocks = DIVUP(num_locs, threads);
  kCrossMapRNorm<<<num_blocks, threads>>>(images->data_device, targets->data_device,
                 num_locs, addScale, powScale, num_filters, sizeF, blocked);
  getLastCudaError("_CrossMapRNorm: kernel execution failed");
}

void _CrossMapRNormUndo(cudamat* outGrads, cudamat* images, cudamat* targets,
                        int num_filters, int sizeF, float addScale,
                        float powScale, bool blocked) {
  int num_locs = (images->size[0] * images->size[1]) / num_filters;
  int threads = 512;
  int batch_offset = 0;

  float *denoms;
  int max_batch_size = ((long) MAX_MEMORY_BYTES) / (sizeof(float) * num_filters);
  max_batch_size = MIN(num_locs, max_batch_size);
  hipError_t err;
  err = hipMalloc((void**)&denoms, max_batch_size * num_filters * sizeof(float));
  if (hipSuccess != err) {
    printf("Out of memory on GPU!\n");
  }
  int num_batches = DIVUP(num_locs, max_batch_size);
  for (int i = 0; i < num_batches; i++) {
    int batch_size = MIN(max_batch_size, num_locs - batch_offset);
    int num_blocks = DIVUP(batch_size, threads);
    kCrossMapDenoms<<<num_blocks, threads>>>(images->data_device, denoms, num_locs, batch_size,
                    batch_offset, addScale, num_filters, sizeF, blocked);

    kCrossMapRNormUndo<<<num_blocks, threads>>>(images->data_device, outGrads->data_device, denoms,
                       targets->data_device, num_locs, batch_size, batch_offset,
                       addScale, powScale, num_filters, sizeF, blocked);
    batch_offset += batch_size;
  }

  hipFree(denoms);
  getLastCudaError("_CrossMapRNormUndo: kernel execution failed");
}

#ifdef __cplusplus
extern "C" {
#endif

void convUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D* images_shape, Shape4D* filters_shape,
                Shape4D* targets_shape, ConvDesc conv_desc,
                float scaleTargets) {
  _convUpGemm(images, filters, targets, *images_shape, *filters_shape,
              *targets_shape, conv_desc, scaleTargets, 1.0, true);
}
void convDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
              Shape4D* derivs_shape, Shape4D* filters_shape,
              Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _convDownGemm(derivs, filters, targets, *derivs_shape, *filters_shape,
                *targets_shape, conv_desc, scaleTargets, 1.0, true);
}

void convOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D* images_shape, Shape4D* derivs_shape, Shape4D* targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convOutpGemm(images, derivs, targets, *images_shape, *derivs_shape,
              *targets_shape, conv_desc, scaleTargets, scaleOutput, true);
}

void localUpGemm(cudamat* images, cudamat* filters, cudamat* targets,
                Shape4D* images_shape, Shape4D* filters_shape,
                Shape4D* targets_shape, ConvDesc conv_desc,
                float scaleTargets) {
  _convUpGemm(images, filters, targets, *images_shape, *filters_shape,
              *targets_shape, conv_desc, scaleTargets, 1.0, false);
}
void localDownGemm(cudamat* derivs, cudamat* filters, cudamat* targets,
              Shape4D* derivs_shape, Shape4D* filters_shape,
              Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _convDownGemm(derivs, filters, targets, *derivs_shape, *filters_shape,
                *targets_shape, conv_desc, scaleTargets, 1.0, false);
}

void localOutpGemm(cudamat* images, cudamat* derivs, cudamat* targets,
              Shape4D* images_shape, Shape4D* derivs_shape, Shape4D* targets_shape,
              ConvDesc conv_desc, float scaleTargets, float scaleOutput) {
  _convOutpGemm(images, derivs, targets, *images_shape, *derivs_shape,
              *targets_shape, conv_desc, scaleTargets, scaleOutput, false);
}

void MaxPoolGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
             Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets, float scaleOutput){
  MaxPooler pooler;
  _convPoolGemm<MaxPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, scaleTargets, scaleOutput, pooler);
}

void AvgPoolGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
             Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets, float scaleOutput){
  AvgPooler pooler;
  _convPoolGemm<AvgPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, scaleTargets, scaleOutput, pooler);
}

void MaxPoolUndoGemm(cudamat* images, cudamat* maxGrads, cudamat* maxActs,
                 cudamat* targets, Shape4D* images_shape, Shape4D* maxGrads_shape,
                 ConvDesc conv_desc, float scaleTargets) {
  _maxPoolUndoGemm(images, maxGrads, maxActs, targets, *images_shape,
                   *maxGrads_shape, conv_desc, scaleTargets, 1);
}

void AvgPoolUndoGemm(cudamat* avgGrads, cudamat* targets, Shape4D* avgGrads_shape,
                 Shape4D* targets_shape, ConvDesc conv_desc, float scaleTargets) {
  _avgPoolUndoGemm(avgGrads, targets, *avgGrads_shape, *targets_shape, conv_desc,
                   scaleTargets, 1);
}

void UpSampleGemm(cudamat* images, cudamat* targets, Shape4D* images_shape,
              Shape4D* targets_shape, int factor, float scaleTargets) { 
  ConvDesc conv_desc;
  conv_desc.kernel_size_y = factor;
  conv_desc.kernel_size_x = factor;
  conv_desc.stride_y = factor;
  conv_desc.stride_x = factor;
  conv_desc.padding_y = 0;
  conv_desc.padding_x = 0;
  conv_desc.num_input_channels = images_shape->shape[3];
  conv_desc.num_output_channels = targets_shape->shape[3];
  conv_desc.num_groups = 1;
  _avgPoolUndoGemm(images, targets, *images_shape, *targets_shape, conv_desc,
                   scaleTargets, factor * factor);
}

void DownSampleGemm(cudamat* images, cudamat* targets, Shape4D* images_shape, Shape4D* targets_shape, int factor) {
  AvgPooler pooler = AvgPooler();
  ConvDesc conv_desc;
  conv_desc.kernel_size_y = factor;
  conv_desc.kernel_size_x = factor;
  conv_desc.stride_y = factor;
  conv_desc.stride_x = factor;
  conv_desc.padding_y = 0;
  conv_desc.padding_x = 0;
  conv_desc.num_input_channels = images_shape->shape[3];
  conv_desc.num_output_channels = targets_shape->shape[3];
  conv_desc.num_groups = 1;
  _convPoolGemm<AvgPooler>(images, targets, *images_shape, *targets_shape,
                           conv_desc, 0, 1, pooler);
}

void ResponseNormCrossMapGemm(
  cudamat* images, cudamat* targets, int num_filters, int sizeF, float addScale,
  float powScale, bool blocked) {
  _CrossMapRNorm(images, targets, num_filters, sizeF, addScale, powScale, blocked);
}

void ResponseNormCrossMapUndoGemm(
  cudamat* outGrads, cudamat* inputs, cudamat* targets, int num_filters,
  int sizeF, float addScale, float powScale, bool blocked) {
  _CrossMapRNormUndo(outGrads, inputs, targets, num_filters, sizeF, addScale,
                     powScale, blocked);
}
#ifdef __cplusplus
}
#endif
